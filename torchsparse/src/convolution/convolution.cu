#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <chrono>
#include <algorithm>
#include "convolution_gpu.h"

template <typename scalar_t>
__global__ void gather_kernel(const int n_k, const int n_in, const int c, 
                               const scalar_t *in_feat, scalar_t *out_feat, const int *kmap,
                               const bool transpose){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / c;
    int j = index % c;
    if(i >= n_k) return;
    int in_pos = kmap[2 * i + transpose];
    if(in_pos < 0) return;
    out_feat[i * c + j] = in_feat[in_pos * c + j];
}

template <typename scalar_t>
__global__ void scatter_kernel(const int n_in, const int n_out, const int c, 
                               const scalar_t *in_feat, scalar_t *out_feat, const int *kmap,
                               const bool transpose){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / c;
    int j = index % c;
    if(i >= n_in) return;
    int out_pos = kmap[2 * i + 1 - transpose];
    if(out_pos < 0) return;
    out_feat[out_pos * c + j] += in_feat[i * c + j];
}

void ConvolutionForwardGPU(at::Tensor in_feat, at::Tensor out_feat,
                           at::Tensor kernel, at::Tensor neighbor_map,
                           at::Tensor neighbor_offset, const bool transpose)
{
    if (in_feat.size(1) != kernel.size(1))
    {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }

    bool is_half = in_feat.scalar_type() == at::ScalarType::Half;

    int out_nrows = out_feat.size(0);
    out_feat.resize_({out_nrows, kernel.size(2)});
    out_feat.zero_();

    int kernel_volume = kernel.size(0);

    // memory optimization
    bool flag = false;
    int in_buffer_size = 1;
    if (kernel_volume % 2 && out_nrows == in_feat.size(0))
    {
        flag = true;
        in_buffer_size = *std::max_element(neighbor_offset.data_ptr<int>(),
                                           neighbor_offset.data_ptr<int>() + kernel_volume / 2);
        in_buffer_size = std::max(in_buffer_size,
                                  *std::max_element(neighbor_offset.data_ptr<int>() + kernel_volume / 2 + 1,
                                                    neighbor_offset.data_ptr<int>() + kernel_volume));
        in_buffer_size = std::max(in_buffer_size, 1);

        torch::mm_out(out_feat, in_feat, kernel[kernel_volume / 2]);
    }
    else
    {
        in_buffer_size = *std::max_element(neighbor_offset.data_ptr<int>(),
                                           neighbor_offset.data_ptr<int>() + kernel_volume);
    }

    auto options =
        torch::TensorOptions().dtype(in_feat.dtype()).device(in_feat.device());
    auto in_buffer = torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto out_buffer = torch::zeros({in_buffer_size, kernel.size(2)}, options);
    int cur_offset = 0;
    for (int i = 0; i < kernel_volume; i++)
    {
        if (flag && (i == kernel_volume / 2))
        {
            cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
            continue;
        }

        if (neighbor_offset.data_ptr<int>()[i] == 0)
        {
            continue;
        }

        at::Tensor out_buffer_activated;
        at::Tensor in_buffer_activated;
        if (is_half)
        {
            out_buffer_activated =
                torch::from_blob(out_buffer.data_ptr<at::Half>(),
                                 {neighbor_offset.data_ptr<int>()[i], kernel.size(2)}, options);
            in_buffer_activated =
                torch::from_blob(in_buffer.data_ptr<at::Half>(),
                                 {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);
        }
        else
        {
            out_buffer_activated =
                torch::from_blob(out_buffer.data_ptr<float>(),
                                 {neighbor_offset.data_ptr<int>()[i], kernel.size(2)}, options);
            in_buffer_activated =
                torch::from_blob(in_buffer.data_ptr<float>(),
                                 {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);
        }

        int n_in = in_buffer_activated.size(0);
        int n_out = in_feat.size(0);
        int c = kernel.size(1);
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(in_feat.type(), "ConvolutionForwardGPU", ([&] {
                                        gather_kernel<scalar_t><<<ceil((double)(n_in * c) / 256), 256>>>(
                                            n_in,
                                            n_out,
                                            c,
                                            in_feat.data_ptr<scalar_t>(),
                                            in_buffer_activated.data_ptr<scalar_t>(),
                                            neighbor_map.data_ptr<int>() + cur_offset,
                                            transpose);
                                    }));

        // gemm
        torch::mm_out(out_buffer_activated, in_buffer_activated, kernel[i]);

        n_in = neighbor_offset.data_ptr<int>()[i];
        n_out = out_nrows;
        c = kernel.size(2);
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(in_feat.type(), "ConvolutionForwardGPU", ([&] {
                                        scatter_kernel<scalar_t><<<ceil((double)(n_in * c) / 256), 256>>>(
                                            neighbor_offset.data_ptr<int>()[i],
                                            out_nrows,
                                            kernel.size(2),
                                            out_buffer_activated.data_ptr<scalar_t>(),
                                            out_feat.data_ptr<scalar_t>(),
                                            neighbor_map.data_ptr<int>() + cur_offset,
                                            transpose);
                                    }));

        cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
    }
}

void ConvolutionBackwardGPU(
    at::Tensor in_feat, at::Tensor grad_in_feat, at::Tensor grad_out_feat,
    at::Tensor kernel, at::Tensor grad_kernel, at::Tensor neighbor_map,
    at::Tensor neighbor_offset, const bool transpose)
{
    grad_in_feat.resize_as_(in_feat);
    grad_in_feat.zero_();
    grad_kernel.resize_as_(kernel);
    grad_kernel.zero_();

    int kernel_volume = kernel.size(0);
    bool flag = false;
    int in_buffer_size;
    in_buffer_size = *std::max_element(neighbor_offset.data_ptr<int>(),
                                       neighbor_offset.data_ptr<int>() + kernel_volume);

    auto options =
        torch::TensorOptions().dtype(in_feat.dtype()).device(in_feat.device());
    auto in_buffer = torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto in_grad_buffer = torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto out_grad_buffer = torch::zeros({in_buffer_size, kernel.size(2)}, options);

    int cur_offset = 0;
    for (int i = 0; i < kernel_volume; i++)
    {
        auto kernel_grad_buffer = grad_kernel[i];
        if (flag && (i == kernel_volume / 2))
        {
            cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
            continue;
        }

        if (neighbor_offset.data_ptr<int>()[i] == 0)
        {
            continue;
        }

        auto out_grad_buffer_activated =
            torch::from_blob(out_grad_buffer.data_ptr<float>(),
                             {neighbor_offset.data_ptr<int>()[i], kernel.size(2)}, options);
        auto in_grad_buffer_activated =
            torch::from_blob(in_grad_buffer.data_ptr<float>(),
                             {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);
        auto in_buffer_activated =
            torch::from_blob(in_buffer.data_ptr<float>(),
                             {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);

        // // gather
        // gather_launch(out_grad_buffer_activated.size(0), grad_out_feat.size(0), kernel.size(2),
        //               grad_out_feat.data_ptr<float>(), out_grad_buffer_activated.data_ptr<float>(),
        //               neighbor_map.data_ptr<int>() + cur_offset, !transpose);

        // gather_launch(in_buffer_activated.size(0), in_feat.size(0), kernel.size(1),
        //               in_feat.data_ptr<float>(), in_buffer_activated.data_ptr<float>(),
        //               neighbor_map.data_ptr<int>() + cur_offset, transpose);

        // gemm
        torch::mm_out(in_grad_buffer_activated, out_grad_buffer_activated, torch::transpose(kernel[i], 0, 1));
        torch::mm_out(kernel_grad_buffer, torch::transpose(in_buffer_activated, 0, 1), out_grad_buffer_activated);

        // // scatter
        // scatter_launch(neighbor_offset.data_ptr<int>()[i], in_feat.size(0), kernel.size(1), in_grad_buffer_activated.data_ptr<float>(),
        //                grad_in_feat.data_ptr<float>(), neighbor_map.data_ptr<int>() + cur_offset, !transpose);

        cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
    }
}
